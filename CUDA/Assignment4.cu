#define BLOCK_DIM	4
#define TILE_DIM	BLOCK_DIM


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<time.h>

void Print_Matrix( int* mtxArray , int n, int m );
void PrintMatrixToText(int* mtxArray, int height, int width, const char* fileName);

// Matrix Mult Kernel
__global__ void matrixMult(int* A, int* B, int* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {
	
	__shared__ int As[TILE_DIM][TILE_DIM];
    __shared__ int Bs[TILE_DIM][TILE_DIM];
    
    int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * blockDim.y + ty;
	int Col = bx * blockDim.x + tx;
	
	int CValue = 0;
	int NUM_PHASE = (int)ceil((float)(ACols + BLOCK_DIM - 1) / (float)BLOCK_DIM);

    for (int k = 0; k < NUM_PHASE; ++k) {
			
        if (k*TILE_DIM + tx < ACols && Row < ARows)	
			As[ty][tx] = A[Row*ACols + k*TILE_DIM + tx];
			
		else													
			As[ty][tx] = 0;

		if (k*TILE_DIM + ty < BRows && Col < BCols)	
			Bs[ty][tx] = B[(k*TILE_DIM + ty)*BCols + Col];
			
		else													
			Bs[ty][tx] = 0;
         
		__syncthreads();

			for (int i = 0; i < TILE_DIM; ++i) 
				CValue += As[ty][i] * Bs[i][tx];
				
		__syncthreads();
    }
    
    if (Row < CRows && Col < CCols) 
		C[(Row*CCols)+Col] = CValue;
		
}

int main(){
	int *h_a, *h_b, *h_c;	// Host Variables
	int *d_a, *d_b, *d_c;	// Device Variables
	int n,m,k;
	
	// Receive Parameters
	/*Todo:Write Receive Parameters through console commands*/
	printf("n = ");
	scanf("%d", &n);
	printf("m = ");
	scanf("%d", &m);
	printf("k = ");
	scanf("%d", &k);
	printf("\n");
	
	// Alloc & Initialize Host Input Matrices	
	int sizeA = (n) * (m) * sizeof(int);
	int sizeB = (m) * (k) * sizeof(int);
	int sizeC = (n) * (k) * sizeof(int);
	h_a = (int*)malloc( sizeA );
	h_b = (int*)malloc( sizeB );
	h_c = (int*)malloc( sizeC );
	
	/*Todo: Initialize A and B with Random Varialbles Ranged (-10, 10)*/
	srand(time(NULL));
		
	for ( int i=0 ; i<n ; i++ )
		for ( int j=0 ; j<m ; j++ )
			h_a[i*m+j] = rand() % 20 - 10;
		
	for ( int i=0 ; i<m ; i++ )
		for ( int j=0 ; j<k ; j++ )
			h_b[i*k+j] = rand() % 20 - 10;

	// Allocate Device Memory
	hipMalloc((void **) &d_a, sizeA);
	hipMalloc((void **) &d_b, sizeB);
	hipMalloc((void **) &d_c, sizeC);

	// Copy Input Matrices to Device Memory
	hipMemcpy(d_a, h_a, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeB, hipMemcpyHostToDevice);

	
	// Set Grid/Block Dimensions
	dim3 dimGrid((int)ceil((float)(BLOCK_DIM+n-1)/(float)BLOCK_DIM), (int)ceil((float)(BLOCK_DIM+k-1)/(float)BLOCK_DIM));
	dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);

	// Launch Kernel
	matrixMult<<<dimGrid ,dimBlock >>>(d_a , d_b , d_c , n, m, m, k, n, k);

	// Copy Result to Host
	hipMemcpy( h_c, d_c, sizeC, hipMemcpyDeviceToHost );
	
	// Free Device Memory
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	// Print Results
	printf("[Input matrix A]\n");	Print_Matrix( h_a , n, m );
	printf("[Input matrix B]\n");	Print_Matrix( h_b , m, k );
	
	printf("[GPU Result]\n");	Print_Matrix( h_c , n, k );
	
	// Print Results
	PrintMatrixToText(h_a,n,m,"matrixA.txt");
	PrintMatrixToText(h_b,m,k,"matrixB.txt");
	PrintMatrixToText(h_c,n,k,"matrixC.txt");
	
	// Free Host Memory
	free(h_a); free(h_b); free(h_c);
	
	return 0;
}

// Utilities
void Print_Matrix( int* mtxArray , int n, int m )
{
	for ( int i=0 ; i<n ; i++ )
	{
		for ( int j=0 ; j<m ; j++ )
		{
			printf( "%d " , mtxArray[i*m+j] );
		}
		printf( "\n" );
	}	
	printf( "\n" );
}

void PrintMatrixToText(int* mtxArray, int height, int width, const char* fileName){
	FILE *f = fopen(fileName, "w");
	for ( int i=0 ; i<height ; i++ )
	{
		for ( int j=0 ; j<width ; j++ )
		{
			fprintf(f, "%d\t" , mtxArray[i*width+j] );
		}
		fprintf(f,"\n" );
	}	
	
	fclose(f);
}

